#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdlib.h>
#include <string.h>

__global__ void mul( float *Ad, float *Bd, float *Cd, int msize, int tile, int task);

int main( int argc, char **argv){

	// argv[0]: name, argv[1]: msize, argv[2]: tile_width/ per block, argv[3]: task per thread, argv[4]: isVerification
	clock_t start = clock();
	int i, j;
	int msize = atoi(argv[1]);
	int tile = atoi(argv[2]);
	int task = atoi(argv[3]);

	float *A, *B, *C;
	float *Ad, *Bd, *Cd;

	A = (float*)malloc(msize * msize * sizeof(float));
	B = (float*)malloc(msize * msize * sizeof(float));
	C = (float*)malloc(msize * msize * sizeof(float));

	// for verification
	if( argc == 5){
		for( i = 0; i < msize; i++){
			for( j = 0; j < msize; j++){
				A[i * msize + j] = (float)1;
				B[i * msize + j] = (float)1;
			}
		}
	}
	else{
		for( i = 0; i < msize; i++){
			for ( j = 0; j < msize; j++){
				srand(time(NULL));
				A[ i * msize + j] = (float)(rand()%2);
				srand(time(NULL));
				B[ i * msize + j] = (float)(rand()%2);
			}
		}
	}

	hipMalloc((void**)&Ad, msize * msize * sizeof(float));
	hipMalloc((void**)&Bd, msize * msize * sizeof(float));
	hipMalloc((void**)&Cd, msize * msize * sizeof(float));
	
	hipMemcpy(Ad, A, msize * msize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Bd, B, msize * msize * sizeof(float), hipMemcpyHostToDevice);

	dim3 dimGrid((msize/tile), (msize/tile));
	dim3 dimBlock((tile/task), (tile/task));

	mul<<<dimGrid, dimBlock>>> (Ad, Bd, Cd, msize, tile, task);

	hipMemcpy(C, Cd, msize * msize * sizeof(float), hipMemcpyDeviceToHost);

	//print out the verification result
	int hoosh = 0;
	if( argc == 5){
		printf("\n=================== V =========================\n");
		for ( i = 0; i < msize; i++){
			printf("\n");
			for( j = 0; j < msize; j++){
				printf("%.2f ", C[i * msize + j]);
				if ( C[i * msize + j] != msize)
					hoosh++;
			}
		}

		printf("\n hoosh = %d \n", hoosh);
		if( hoosh != 0 )
			printf("\n YOU JUST DRIVING ME CRAZZZZZZZZZZZZZZZZZZZZZZZZZZZZZZZZZZZZZZZZZZZZZY\n");
		else
			printf("\n GOOD! ONLY ONE MORE TO GO!!! BIG BIG : ) \n");

	}

	free(A); free(B); free(C);
	hipFree(Ad);hipFree(Bd);hipFree(Cd);

	printf( " \n msize: %d\t tilewidth: %d\t task: %d\t timeElapsed: %f\n", msize, tile, task, ((double)(clock()-start)/CLOCKS_PER_SEC));
	return 1;
}

__global__ void mul( float *Ad, float *Bd, float *Cd, int msize, int tile, int task){

	int tx, ty;
	int r, c;
	float Cv;
	int m;

	for ( tx = 0; tx < task; tx++){
		for ( ty = 0; ty < task; ty++){
			r = blockIdx.y * tile + threadIdx.y * task + ty;
			c = blockIdx.x * tile + threadIdx.x * task + tx;
			Cv = (float)0;
			for ( m = 0; m < msize; m++){
				Cv += Ad[ r * msize + m] * Bd[ m * msize + c];
			}
			Cd[ r * msize + c] = Cv;
		}
	}
}	
