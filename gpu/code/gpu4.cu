#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdlib.h>
#include <string.h>


__global__ void mul( float *Ad, float *Bd, float *Cd, int msize, int tile, int task);

int main( int argc, char **argv){

	// argv[0]: name, argv[1]: msize, argv[2]: tile_width/ per block, argv[3]: task per thread, argv[4]: isVerification
	clock_t start = clock();
	int i, j;
	int msize = atoi(argv[1]);
	int tile = atoi(argv[2]);
	int task = atoi(argv[3]);

	float *A, *B, *C;
	float *Ad, *Bd, *Cd;

	A = (float*)malloc(msize * msize * sizeof(float));
	B = (float*)malloc(msize * msize * sizeof(float));
	C = (float*)malloc(msize * msize * sizeof(float));

	// for verification
	if( argc == 5){
		for( i = 0; i < msize; i++){
			for( j = 0; j < msize; j++){
				A[i * msize + j] = (float)1;
				B[i * msize + j] = (float)1;
			}
		}
	}
	else{
		for( i = 0; i < msize; i++){
			for ( j = 0; j < msize; j++){
				srand(time(NULL));
				A[ i * msize + j] = (float)(rand()%2);
				srand(time(NULL));
				B[ i * msize + j] = (float)(rand()%2);
			}
		}
	}

	hipMalloc((void**)&Ad, msize * msize * sizeof(float));
	hipMalloc((void**)&Bd, msize * msize * sizeof(float));
	hipMalloc((void**)&Cd, msize * msize * sizeof(float));
	
	hipMemcpy(Ad, A, msize * msize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Bd, B, msize * msize * sizeof(float), hipMemcpyHostToDevice);

	dim3 dimGrid((msize/tile), (msize/tile));
	dim3 dimBlock((tile/task), (tile/task));

	size_t SharedMemBytes = 2 * tile * msize * sizeof(float);
	mul<<<dimGrid, dimBlock, SharedMemBytes>>> (Ad, Bd, Cd, msize, tile, task);

	hipMemcpy(C, Cd, msize * msize * sizeof(float), hipMemcpyDeviceToHost);

	//print out the verification result
	int beZero = 0;
	if( argc == 5){
		printf("\n=================== V =========================\n");
		for ( i = 0; i < msize; i++){
			printf("\n");
			for( j = 0; j < msize; j++){
				printf("%.2f ", C[i * msize + j]);
				if ( C[i * msize + j] != msize)
					beZero++;
			}
		}

		if( beZero != 0 )
			printf("\n ==========Told you to be steady MAN!=============\n");
		else
			printf("\n GOOD! ONLY ONE MORE TO GO!!! BIG BIG : ) \n");

	}

	free(A); free(B); free(C);
	hipFree(Ad);hipFree(Bd);hipFree(Cd);

	printf( " \n msize: %d\t tilewidth: %d\t task: %d\t timeElapsed: %f\n", msize, tile, task, ((double)(clock()-start)/CLOCKS_PER_SEC));
	return 1;
}

__global__ void mul( float *Ad, float *Bd, float *Cd, int msize, int tile, int task){

	extern __shared__ float shared[];// first half is for Shared A, second half is for Shared B
	int tx, ty;
	int r, c;
	float Cv;
	float Av, Bv;
	int m;

	for ( tx = 0; tx < task; tx++){
		c = blockIdx.x * tile + threadIdx.x * task + tx;
		for ( ty = 0; ty < task; ty++){
			r = blockIdx.y * tile + threadIdx.y * task + ty;
			Cv = (float)0;
			Av = Ad[ r * msize]; // initialize
			Bv = Bd[ c ];
			for ( m = 0; m < msize; m++){
				shared[ threadIdx.y * task + ty] = Av; // put cur tile to shared mem
				shared[ tile * msize + threadIdx.x * task + tx] = Bv;
				__syncthreads();
				if( (m + 1) < msize){
					Av = Ad[ r * msize + m + 1]; //load next tile to reg
					Bv = Bd[ m * msize + c];
				}
				Cv += shared[ threadIdx.y * task + ty ] * shared[ tile * msize + threadIdx.x * task + tx];
				__syncthreads();
			}
			Cd[ r * msize + c] = Cv;
		}
	}
}	
