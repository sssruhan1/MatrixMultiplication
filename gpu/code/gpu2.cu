#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

__global__  void mul(float *Ad, float *Bd, float *Cd, int msize, int tilewidth);

int main( int argc, char **argv){
	
	clock_t start = clock();
	int i, j;
	int tile;
	int msize;
	
	msize = atoi(argv[1]); //matrix size 
	tile = atoi(argv[2]); //tile width

	float *A, *B, *C;

	A = (float*)malloc(msize * msize * sizeof(float));
	B = (float*)malloc(msize * msize * sizeof(float));
	C = (float*)malloc(msize * msize * sizeof(float));

		
	if (argc == 4) { // for verification
		for( i = 0; i < msize; i++){
			for ( j = 0; j < msize; j++){
				A[ i * msize + j ] = (float)1;
				B[ i * msize + j ] = (float)1;
			}
		}
	}
	
	else{
		for( i = 0; i < msize; i++){
			for( j = 0; j < msize; j++){
				srand( time(NULL));
				A[ i * msize + j] = (float)(rand()%2);
				srand( time(NULL));
				B[ i * msize + j] = (float)(rand()%2);
			}
		}
	}

	float *Ad, *Bd, *Cd;
	
	hipMalloc( (void**)&Ad, msize * msize * sizeof(float));
	hipMalloc( (void**)&Bd, msize * msize * sizeof(float));
	hipMalloc( (void**)&Cd, msize * msize * sizeof(float));

	hipMemcpy( Ad, A, msize * msize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy( Bd, B, msize * msize * sizeof(float), hipMemcpyHostToDevice);

	dim3 dimGrid( (msize/tile), (msize/tile));
	dim3 dimBlock( tile, tile);

	mul<<<dimGrid, dimBlock>>>( Ad, Bd, Cd, msize, tile);

	hipMemcpy( C, Cd, msize * msize * sizeof(float), hipMemcpyDeviceToHost);
	
	// verify the result
	int knife = 0;
	if ( argc == 4){
		printf("\n ================== V ======================\n");
		for ( i = 0; i < msize; i++){
			printf("\n");
			for ( j = 0; j < msize; j++){
				printf("%f ", C[ i * msize + j]);
				if ( C[ i * msize + j] != msize)
					knife ++;
			}
		}
		if ( knife != 0) 
			printf("\n ================= OH CRAP! ====================\n");
		else
			printf("\n ================= MO HA HA : ) ====================\n");

	}
	free(A); free(B); free(C);
	hipFree(Ad); hipFree(Bd); hipFree(Cd);

	printf("\n msize: %d\t tilewidth: %d\t time elapsed: %f\n", msize, tile, ((double)(clock() - start))/CLOCKS_PER_SEC); 
	return 1;
}

__global__ void mul( float *Ad, float *Bd, float *Cd, int msize, int tilewidth){

	int r = blockIdx.y * tilewidth + threadIdx.y;
	int c = blockIdx.x * tilewidth + threadIdx.x;

	float Cv = 0;
	int i;
	for( i = 0; i < msize; i++)
		Cv += Ad[ r * msize + i] * Bd[ c + i * msize];

	Cd[ r * msize + c] = Cv;
}


