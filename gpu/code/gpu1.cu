#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <stdlib.h>

__global__ void mul(float* Ad, float* Bd, float* Cd, int msize);

int main(int argc, char **argv){

	clock_t start = clock();
	int msize;
	msize = atoi(argv[1]);
	int i, j;
	
	//input matrix	
	float *A,*B,*C; // matrix A, B and C
	A = (float*)malloc(msize*msize*sizeof(float));
	B = (float*)malloc(msize*msize*sizeof(float));
	C = (float*)malloc(msize*msize*sizeof(float));
        
	//for verification
        if (argc == 3){
		printf("\n ==============Verifying===================\n"); 
		for ( i = 0; i < msize; i++){
			for ( j = 0; j < msize; j++){
				A[i * msize + j ] = (float)1;
				B[i * msize + j ] = (float)1;
			}
		}
	}

	else{
		for (i = 0; i < msize; i++) {
			for (j = 0; j < msize; j++) {
				srand(time(NULL));
				A[i * msize + j] = (float)(rand()%2);
	    			srand(time(NULL));
	    			B[i * msize + j] = (float)(rand()%2);
			}
		}
	}

	float  *Ad;
	float  *Bd;
	float  *Cd;


	hipMalloc((void**)&Ad, msize * msize * sizeof(float));
	hipMalloc((void**)&Bd, msize * msize * sizeof(float));
	hipMalloc((void**)&Cd, msize * msize * sizeof(float));

	hipMemcpy( Ad, A, msize * msize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy( Bd, B, msize * msize * sizeof(float), hipMemcpyHostToDevice);
	
	dim3 dimGrid(1, 1);
	dim3 dimBlock(msize, msize);

	mul<<<dimGrid, dimBlock>>> ( Ad, Bd, Cd, msize);
				
	hipMemcpy( C, Cd, msize * msize * sizeof(float), hipMemcpyDeviceToHost);

	// print the verification result
	int Meow = 0;
	if ( argc == 3){
		for ( i = 0; i < msize; i++){
			printf("\n");
			for ( j = 0; j < msize; j++){
				printf("%d ", (int)C[i * msize + j]);
				if( C[ i * msize + j] != msize){
					Meow++;
				}
			}
		}
		if( Meow > 0 )
			printf("\n\n==========Ding, ding, ding! Please don't kill yourself! Calm down, buddy!==========\n");
		else
			printf("\n\n=========================Finally! You got it! BIG :-) =============================\n");
	}

	free(A);
	free(B);
	free(C);
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);

	printf("\n msize: %d \n TIME ELAPSED : %f \n", msize, ((double)(clock()-start))/CLOCKS_PER_SEC);
	return 1;
}

__global__ void mul(float* Ad, float* Bd, float* Cd, int msize){

	float Avalue, Bvalue;
	float Cvalue = 0;
	
	int i;
	for ( i = 0; i < msize; i++){	
		Avalue = Ad[threadIdx.y * msize + i];
		Bvalue = Bd[i * msize + threadIdx.x];
		Cvalue += Avalue * Bvalue;
	}
	
	Cd[threadIdx.y * msize + threadIdx.x] = Cvalue;
	//Cd[threadIdx.y] = 1;
}
 
