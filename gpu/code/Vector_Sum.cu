#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 10

__global__ void add( int *a, int *b, int *c) {
	int tid = blockIdx.x;	//Handle the data at the index
	if (tid < N)
		c[tid] = a[tid] + b[tid];
}


int main(){
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	int i;
	
	//allocate the memory on the GPU
	hipMalloc ( (void**)&dev_a, N * sizeof (int) );
	hipMalloc ( (void**)&dev_b, N * sizeof (int) );
	hipMalloc ( (void**)&dev_c, N * sizeof (int) );
	
	//fill the arrays 'a' and 'b' on the CPU
	for ( i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i*i;
	}
	
	printf("Vector a is :\n");
	for(i=0; i<N; i++) printf("%d  ", a[i]);
	printf("\nVector b is :\n");
	for(i=0; i<N; i++) printf("%d  ", b[i]);
	
	//copy the arrays 'a' and 'b' to the GPU
	hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
								
	add<<<N, 1>>> (dev_a, dev_b, dev_c);
	
	//copy the array 'c' back from the GPU to the CPU
	hipMemcpy( c, dev_c, N * sizeof(int),hipMemcpyDeviceToHost );
								
								
	//display the results
	printf("\nVector c = a+b:\n");
	for ( i = 0; i < N; i++) {
//		printf( "%d + %d = %d\n", a[i], b[i],c[i]);
		printf( "%d  ", c[i]);  
	}
	printf("\n");	
	
	//free the memory allocated on the GPU
	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );
	
	return 0;
}





	
